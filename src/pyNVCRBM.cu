extern "C"{
#include <Python.h>
}
#include <iostream>
#include "matrix.h"
#include "nvmatrix.cuh"
#include "crbm.cuh"
#include <hip/hip_runtime.h>
#include <ctime>
#include <cstdlib>
#include "utils.h"

using namespace std;

static CRBM* crbm = NULL;

static PyArrayObject* copy_host_matrix(Matrix& src){
    PyArrayObject* ret;
    int dims[2];

    dims[0] = src.get_row_num();
    dims[1] = src.get_col_num();
    ret = (PyArrayObject*)PyArray_FromDims(2, dims, NPY_FLOAT);

    memcpy(ret->data, src.get_data(), sizeof(float) * src.get_ele_num());
    
    return ret;
}

static PyObject*
init(PyObject *self, PyObject *args){
    PyArrayObject *pyfilter;
    PyArrayObject *pyinit_filter, *pyinit_hbias, *pyinit_vbias;
    int filter_num;
    int filter_size;
    int input_num;
    int input_size;
    int input_group_num;
    int pooling_rate;
    int left_upper_padding, right_low_padding;

    if(!PyArg_ParseTuple(args, "iiiiiiiiO!O!O!", 
                &filter_num, &filter_size,
                &input_num, &input_size,
                &input_group_num, &pooling_rate,
                &left_upper_padding, &right_low_padding,
                &PyArray_Type, &pyinit_filter,
                &PyArray_Type, &pyinit_hbias,
                &PyArray_Type, &pyinit_vbias))
        return NULL;

    Matrix init_filter(pyinit_filter);
    Matrix init_hbias(pyinit_hbias);
    Matrix init_vbias(pyinit_vbias);

    crbm = new CRBM(filter_num, filter_size,
              input_num, input_size, input_group_num,
              left_upper_padding, right_low_padding,
              pooling_rate, &init_filter, //&filter,
              &init_hbias, &init_vbias);

    pyfilter = copy_host_matrix(*crbm->CPU_filters);

    return PyArray_Return(pyfilter);
    //return Py_BuildValue("i", 0);
}

static PyObject*
run_batch(PyObject *self, PyObject *args){
    PyArrayObject *pybatch_data;
    int cur_trail, cur_image, cur_batch;

    if(!PyArg_ParseTuple(args, "iiiO!", 
        &cur_trail, &cur_image, &cur_batch,
        &PyArray_Type, &pybatch_data)){
        return NULL;
    }
    Matrix batch_data(pybatch_data);

    crbm->run_batch(cur_trail, cur_image, cur_batch, batch_data);
    
    return Py_BuildValue("i", 0);
}

static PyObject*
get_gpu_filters(PyObject *self, PyObject *args){
    PyArrayObject *pyfilter;

    Matrix *tmp_filter = new Matrix(*crbm->CPU_filters);
    crbm->GPU_filters->assign(*tmp_filter);
    pyfilter = copy_host_matrix(*tmp_filter);
    delete tmp_filter;

    return PyArray_Return(pyfilter);
}

static PyObject*
get_cpu_filters(PyObject *self, PyObject *args){
    PyArrayObject *pyfilter;

    pyfilter = copy_host_matrix(*crbm->CPU_filters);

    return PyArray_Return(pyfilter);
}

static PyObject*
get_gpu_hbias(PyObject *self, PyObject *args){
    PyArrayObject *pyhbias;

    Matrix *tmp_hbias = new Matrix(*crbm->CPU_hbias);
    crbm->GPU_hbias->assign(*tmp_hbias);
    pyhbias = copy_host_matrix(*tmp_hbias);
    delete tmp_hbias;

    return PyArray_Return(pyhbias);
}

static PyObject*
print_result(PyObject *self, PyObject *args){
    cout << "ferr : " << crbm->ferr / 20.0 << endl;
    cout << "sparsity : " << crbm->sparsity / 20.0 << endl;
    crbm->ferr = 0.0;
    crbm->sparsity = 0.0;
    return Py_BuildValue("i", 0);
}

static PyMethodDef PyNVcrbmMethods[] = {
    {"get_gpu_filters", get_gpu_filters, METH_VARARGS, "Get the filter weight matrix"},
    {"get_cpu_filters", get_cpu_filters, METH_VARARGS, "Get the filter weight matrix"},
    {"get_gpu_hbias", get_gpu_hbias, METH_VARARGS, "Get gpu hidden layer bias"},
    {"print_result", print_result, METH_VARARGS, "print result"},
    {"run_batch", run_batch, METH_VARARGS, "Run a batch"},
    {"init", init, METH_VARARGS, "Initialize the convolutional RBM"},
    {NULL, NULL, 0, NULL}
};

PyMODINIT_FUNC
initnvcrbm(void){
    (void)Py_InitModule("nvcrbm", PyNVcrbmMethods);
    _import_array();
    srand(1234);
    //srand(time(NULL));
}
