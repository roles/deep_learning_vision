#include "nvmatrix.cuh"
#include <hip/hip_runtime.h>

using namespace std;

NVMatrix::NVMatrix(Matrix &m){
    this->nrow  = m.get_row_num();
    this->ncol  = m.get_col_num();
    this->own   = true; 
    this->trans = m.get_trans();
    hipMalloc((void**)&this->data, nrow * ncol * sizeof(float));
    hipMemcpy(this->data, m.get_data(), nrow * ncol * sizeof(float),
            hipMemcpyHostToDevice);
}

NVMatrix::~NVMatrix(){
    hipFree(this->data);
}

float* NVMatrix::get_data(){
    return this->data;
}

int NVMatrix::get_row_num(){
    return this->nrow;
}

int NVMatrix::get_col_num(){
    return this->ncol;
}

bool NVMatrix::get_trans(){
    return this->trans;
}

void NVMatrix::assign(Matrix &target){
    assert(this->nrow == target.get_row_num() 
        && this->ncol == target.get_col_num());
    hipMemcpy(target.get_data(), this->data, nrow * ncol * sizeof(float),
            hipMemcpyDeviceToHost);
}
