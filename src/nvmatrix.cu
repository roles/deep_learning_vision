#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cmath>
#include "nvmatrix.cuh"
#include "nvmatrix_kernel.cuh"

using namespace std;

NVMatrix::NVMatrix(Matrix &m){
    this->nrow  = m.get_row_num();
    this->ncol  = m.get_col_num();
    this->own   = true; 
    this->trans = m.get_trans();
    hipMalloc((void**)&this->data, nrow * ncol * sizeof(float));
    hipMemcpy(this->data, m.get_data(), nrow * ncol * sizeof(float),
            hipMemcpyHostToDevice);
}

NVMatrix::NVMatrix(int nrow, int ncol){
    this->nrow  = nrow;
    this->ncol  = ncol;
    this->own   = true; 
    this->trans = false;
    hipMalloc((void**)&this->data, nrow * ncol * sizeof(float));
    _init_mat<<<ceil(nrow*ncol/64.0), 64>>>(this->get_data(), 0.0f, nrow*ncol);
    hipDeviceSynchronize();
}

NVMatrix::~NVMatrix(){
    hipFree(this->data);
}

float* NVMatrix::get_data(){
    return this->data;
}

int NVMatrix::get_row_num(){
    return this->nrow;
}

int NVMatrix::get_col_num(){
    return this->ncol;
}

int NVMatrix::get_ele_num(){
    return this->ncol * this->nrow;
}

bool NVMatrix::get_trans(){
    return this->trans;
}

void NVMatrix::assign(Matrix &target){
    assert(this->nrow == target.get_row_num() 
        && this->ncol == target.get_col_num());
    hipMemcpy(target.get_data(), this->data, nrow * ncol * sizeof(float),
            hipMemcpyDeviceToHost);
}

void NVMatrix::ele_scale(float scaler, NVMatrix& target){
    int len = nrow * ncol;
    _ele_scale<<<ceil(len / 64.0), 64>>>(this->get_data(), target.get_data(),
            scaler, len);
    hipDeviceSynchronize();
}

void NVMatrix::ele_scale(float scaler){
    ele_scale(scaler, *this);
}

void NVMatrix::mat_sum(int axis, NVMatrix& target){
    if(axis == 1){      //column sum
        dim3 blocks = dim3(ceil(nrow / 64.0), 1);
        dim3 threads = dim3(64, 1);
        _mat_sum_col<<<blocks, threads>>>(get_data(), target.get_data(), nrow, ncol);
        hipDeviceSynchronize();
    }else{              //row sum
        /*
        dim3 blocks = dim3(ceil(ncol / 64.0), 1);
        dim3 threads = dim3(64, 1);
        _mat_sum_row<<<blocks, threads>>>(get_data(), target.get_data(), nrow, ncol);
        */
        
        int cur_col = ncol;
        NVMatrix* cur_sum_mat = this;
        while(cur_col > 1){
            int agg_col = ceil(cur_col * 1.0 / NUM_THREAD_PER_ROW);
            NVMatrix* agg_sum_mat = new NVMatrix(nrow,  agg_col);

            dim3 blocks = dim3(agg_col, nrow);
            dim3 threads = dim3(NUM_THREAD_PER_ROW, 1);

            _mat_sum_row_fast<<<blocks, threads>>>(cur_sum_mat->get_data(), agg_sum_mat->get_data(),
                nrow, cur_col, agg_col);

            if(cur_sum_mat != this)
                delete cur_sum_mat;
            delete agg_sum_mat;

            cur_sum_mat = agg_sum_mat;
            cur_col     = agg_col;
        }
        _copy_mat<<<ceil(nrow/64.0), 64>>>(cur_sum_mat->get_data(), target.get_data(), 
                cur_sum_mat->get_ele_num());

        if(cur_sum_mat != this)
            delete cur_sum_mat;
        hipDeviceSynchronize();
    }
}
