#include "hip/hip_runtime.h"
#include "matrix.h"
#include "crbm.cuh"
#include <iostream>
#include "utils.h"
#include "crbm_kernel.cuh"

using namespace std;

__global__ void setup_curand_kernel(hiprandState *state, int count){
    int id = threadIdx.x + blockIdx.x * 64;
    if(id < count){
        hiprand_init(1234, id, 0, &state[id]);
    }
}

void setup_curand(hiprandState **state, int count){
    CUDA_CALL(hipMalloc((void**)state, count * sizeof(hiprandState)));
    setup_curand_kernel<<< ceil(count/64.0), 64>>>(*state, count);
}

CRBM::CRBM(int filter_num, int filter_size,
        int input_num, int input_size, int channel_num,
        int left_upper_padding, int right_low_padding,
        int pooling_rate, 
        Matrix *filters, Matrix *vbias,
        Matrix *hbias, Matrix *input){
    this->filter_num    = filter_num;  
    this->filter_size   = filter_size;
    this->input_num     = input_num;
    this->input_size    = input_size; 
    this->pooling_rate  = pooling_rate;
    this->channel_num = channel_num;
    this->left_upper_padding = left_upper_padding;
    this->right_low_padding = right_low_padding;
    this->feature_map_size = input_size + left_upper_padding +
        right_low_padding - filter_size + 1;
    this->subsample_size = feature_map_size / pooling_rate;

    if(filters == NULL){
        this->CPU_filters = filter_init(filter_size, filter_num, channel_num); 
    }else{
        this->CPU_filters = filters;
    }

    if(hbias == NULL){
        this->CPU_hbias = new Matrix(filter_num, 1);
    }else{
        this->CPU_hbias = hbias;
    }

    if(vbias == NULL){
        this->CPU_vbias = new Matrix(channel_num, 1);
    }else{
        this->CPU_vbias = vbias;
    }
    this->CPU_input = input;
    this->CPU_y_h = new Matrix(input_num ,
            filter_num * feature_map_size * feature_map_size);
    this->CPU_y_h_probs = new Matrix(input_num ,
            filter_num * feature_map_size * feature_map_size);
            //filter_num * feature_map_size * feature_map_size, 1, 1);
    this->CPU_y_p = new Matrix(input_num, 
            filter_num * subsample_size * subsample_size);
    this->CPU_y_v = new Matrix(this->CPU_input->get_row_num(),
            this->CPU_input->get_col_num());
    this->CPU_y_v_probs = new Matrix(this->CPU_input->get_row_num(),
            this->CPU_input->get_col_num());
    this->CPU_d_w = new Matrix(this->CPU_filters->get_row_num(),
            this->CPU_filters->get_col_num());

    this->GPU_filters = new NVMatrix(*this->CPU_filters);
    this->GPU_hbias = new NVMatrix(*this->CPU_hbias);
    this->GPU_vbias = new NVMatrix(*this->CPU_vbias);
    this->GPU_input = new NVMatrix(*this->CPU_input);
    this->GPU_y_h = new NVMatrix(*this->CPU_y_h);
    this->GPU_y_h_probs = new NVMatrix(*this->CPU_y_h_probs);
    this->GPU_y_p = new NVMatrix(*this->CPU_y_p);
    this->GPU_y_v = new NVMatrix(*this->CPU_y_v);
    this->GPU_y_v_probs = new NVMatrix(*this->CPU_y_v_probs);
    this->GPU_d_w = new NVMatrix(*this->CPU_d_w);
    this->rnd_state_num = 1;
    setup_curand(&this->rnd_state, this->rnd_state_num);
}

CRBM::~CRBM(){
    delete this->CPU_filters;
    delete this->CPU_hbias;
    delete this->CPU_vbias;
    delete this->CPU_y_h;
    delete this->CPU_y_h_probs;
    delete this->CPU_y_p;
    delete this->CPU_y_v;
    delete this->CPU_y_v_probs;
    delete this->CPU_d_w;

    delete this->GPU_filters;
    delete this->GPU_hbias;
    delete this->GPU_vbias;
    delete this->GPU_y_h;
    delete this->GPU_y_h_probs;
    delete this->GPU_y_p;
    delete this->GPU_y_v;
    delete this->GPU_y_v_probs;
    delete this->GPU_d_w;

    hipFree(this->rnd_state);
}

Matrix* CRBM::filter_init(int filter_size, int filter_num, int channel_num){
    float low   = - 4 * sqrt(6.0 / (2 * filter_size * filter_size * channel_num)); 
    float upper = -low;
    return new Matrix(filter_num, channel_num*filter_size*filter_size, low, upper);
}

void CRBM::CPU_convolution_forward(float *input, float *filter, float *target, float *hbias){

    bzero(target, input_num * filter_num * feature_map_size * feature_map_size * sizeof(float));

    for(int img = 0; img < input_num; img++){
        for(int fil = 0; fil < filter_num; fil++){

            float *curBias = hbias + fil;

            for(int r = 0; r < feature_map_size; r++){
                for(int c = 0; c < feature_map_size; c++){

                    float *curFilter = filter + fil * channel_num * filter_size * filter_size;

                    float* curTarget = target + img * filter_num * feature_map_size * feature_map_size +
                        fil * feature_map_size * feature_map_size +
                        r * feature_map_size + c;

                    for(int k = 0; k < channel_num; k++){

                        float* curInput = input + img * channel_num * input_size * input_size +
                            k * input_size * input_size + 
                            (r < left_upper_padding ? 0 : r - left_upper_padding) * input_size +
                            (c < left_upper_padding ? 0 : c - left_upper_padding);

                        for(int i = 0; i < filter_size; i++){

                            if(!((r+i) < left_upper_padding || 
                                        (r+i) >= (left_upper_padding + input_size))){

                                int step = 0;

                                for(int j = 0; j < filter_size; j++){ 
                                    if(!((c+j) < left_upper_padding ||
                                                (c+j) >= (left_upper_padding + input_size))){
                                        *curTarget += curFilter[i*filter_size+j] * (*curInput);
                                        curInput++;
                                        step++;
                                    }
                                }
                                curInput += input_size - step;

                            }
                        }
                        curFilter += filter_size * filter_size;
                    }
                    *curTarget += *curBias;
                }
            }
        }
    }
}

static int max_pooling_multinomial(float *probs, int len){
    float rnd = random_float(0, 1);
    int i;

    for(i = 0; rnd > probs[i]; i++, probs[i] += probs[i-1]);

    return i;
}

void CRBM::CPU_max_pooling(float *y_h, float *y_h_probs, float *y_p){
    float pooling_area[MAX_POOLING_RATE*MAX_FILETER_SIZE+1];

    for(int img = 0; img < input_num; img++){
        for(int fil = 0; fil < filter_num; fil++){
            float *fm = y_h + 
                img * filter_num * feature_map_size * feature_map_size +
                fil * feature_map_size * feature_map_size;
            float *probs = y_h_probs + 
                img * filter_num * feature_map_size * feature_map_size +
                fil * feature_map_size * feature_map_size;
            float *target = y_p +
                img * filter_num * subsample_size * subsample_size +
                fil * subsample_size * subsample_size;

            for(int i = 0; i < feature_map_size; i += pooling_rate){
                for(int j = 0; j < feature_map_size; j += pooling_rate){

                    float sum = 0;
                    for(int pi = 0; pi < pooling_rate; pi++){
                        for(int pj = 0; pj < pooling_rate; pj++){
                            float *cur_fm = fm + (i+pi) * feature_map_size + (j+pj);
                            *cur_fm = expf(*cur_fm);
                            sum += *cur_fm;
                        }
                    }
                    for(int pi = 0; pi < pooling_rate; pi++){
                        for(int pj = 0; pj < pooling_rate; pj++){
                            float *cur_fm = fm + (i+pi) * feature_map_size + (j+pj);
                            float *cur_probs = probs + (i+pi) * feature_map_size + (j+pj);
                            *cur_probs = *cur_fm / (1 +sum);
                            pooling_area[pi*pooling_rate+pj] = *cur_probs;
                            *cur_fm = 0;
                        }
                    }
                    pooling_area[pooling_rate*pooling_rate] = 1.0/(1+sum);
                    int pooling_idx = max_pooling_multinomial(pooling_area, 
                            pooling_rate*pooling_rate+1);
                    if(pooling_idx == pooling_rate*pooling_rate){
                        target[(i/pooling_rate)*subsample_size+(j/pooling_rate)] = 1;
                    }else{
                        target[(i/pooling_rate)*subsample_size+(j/pooling_rate)] = 0;
                        int pi = pooling_idx / pooling_rate;
                        int pj = pooling_idx % pooling_rate;
                        fm[(i+pi) * feature_map_size + (j+pj)] = 1;
                    }
                }
            }
        }
    }
}

void CRBM::CPU_convolution_backward(float *y_h, float *filters, float *vbias,
        float *y_v_probs, float *y_v){
    float tmp_recon[MAX_IMGAG_SIZE][MAX_IMGAG_SIZE];
    int padding = filter_size-1;
    int input_padding_size = feature_map_size + filter_size - 1;
    int lu_padding = left_upper_padding;

    bzero(tmp_recon, sizeof(tmp_recon));

    for(int img = 0; img < input_num; img++){
        for(int cha = 0; cha < channel_num; cha++){
            float *target = y_v_probs +
                img * channel_num * input_size * input_size +
                cha * input_size * input_size;

            for(int fil = 0; fil < filter_num; fil++){
                float *filter = filters +
                    fil * filter_size * filter_size * channel_num +
                    cha * filter_size * filter_size;

                float *fm = y_h +
                    img * filter_num * feature_map_size * feature_map_size +
                    fil * feature_map_size * feature_map_size;

                for(int r = 0; r < feature_map_size + filter_size - 1; r++){
                    for(int c = 0; c < feature_map_size + filter_size - 1; c++){

                        for(int i = r; i < r+filter_size; i++){
                            for(int j = c; j < c+filter_size; j++){
                                if(!(i < padding || j < padding ||
                                            i >= (padding + feature_map_size) ||
                                            j >= (padding + feature_map_size))){
                                    tmp_recon[r][c] += 
                                        fm[(i-padding)*feature_map_size + (j-padding)] *
                                        filter[(filter_size-1-(i-r))*filter_size + (filter_size-1-(j-c))];
                                }
                            }
                        }
                    }
                }
            }

            for(int i = 0; i < input_size; i++){
                for(int j = 0; j < input_size; j++){
                    target[i*input_size+j] = logisitc(tmp_recon[i+lu_padding][j+lu_padding]);
                    //target[i*input_size+j] = expf(-tmp_recon[i+lu_padding][j+lu_padding]);
                }
            }
            bzero(tmp_recon, sizeof(tmp_recon));
        }
    }
}

/*
 * 分为positive phase和negative phase
 * is_init为true则计算positive phase, dw初始化为0
 * is_init为false则计算negative phase, dw -= new_dw
 */
void CRBM::CPU_compute_d_w(float *v, float *h, float *dw, bool is_init){

    float sign;
    int lu_padding = left_upper_padding;
    if(is_init){
        bzero(dw, filter_num * channel_num * filter_size * filter_size * sizeof(float));
        sign = 1.0f;
    }else{
        sign = -1.0f;
    }

    for(int img = 0; img < input_num; img++){
        for(int fil = 0; fil < filter_num; fil++){

            float *this_h = h + img * filter_num * feature_map_size * feature_map_size +
                fil * feature_map_size * feature_map_size;

            for(int cha = 0; cha < channel_num; cha++){

                float *this_v = v + img * channel_num * input_size * input_size +
                    cha * input_size * input_size;

                float *this_dw = dw + fil * channel_num * filter_size * filter_size +
                    cha * filter_size * filter_size;

                for(int r = 0; r < filter_size; r++){
                    for(int c = 0; c < filter_size; c++){

                        float *cur_v = this_v + (r-lu_padding) * input_size +
                            (c-lu_padding);

                        for(int i = 0; i < feature_map_size; i++){
                            for(int j = 0; j < feature_map_size; j++){
                                if(!((r+i) < lu_padding ||
                                            (c+j) < lu_padding ||
                                            (r+i) >= (lu_padding+input_size) ||
                                            (c+j) >= (lu_padding+input_size))){

                                    this_dw[r*filter_size+c] += 
                                        sign * cur_v[j] * this_h[i*feature_map_size+j];
                                }
                            }
                            cur_v += input_size;
                        }
                    }
                }
            }
        }
    }
}

void CRBM::GPU_convolution_forward(float *input, float *filters, float *y_h, float *hbias){
    dim3 blocks = dim3(input_size / 32 * filter_num, input_size / 32 * input_num);
    dim3 threads = dim3(32, 32);
    convolution_forward_kernel<<<blocks, threads>>>(input, filters, y_h, 
            hbias, input_size, channel_num, feature_map_size, filter_size, 
            filter_num, left_upper_padding);
    hipDeviceSynchronize();
}

void CRBM::GPU_max_pooling(float *y_h, float *y_h_probs, float *y_p){
    dim3 blocks = dim3(feature_map_size / pooling_rate / 16 * filter_num, 
            feature_map_size / pooling_rate / 16 * input_num);
    dim3 threads = dim3(16, 16);
    max_pooling_kernel<<<blocks, threads>>>(y_h, y_h_probs, y_p,
            feature_map_size, filter_num, pooling_rate, rnd_state, rnd_state_num);
    hipDeviceSynchronize();
}

void CRBM::GPU_convolution_backward(float *y_h, float *filters, float *vbias, 
        float *y_v_probs, float *y_v){
    dim3 blocks = dim3(input_size / 32 * channel_num, input_size / 32 * input_num);
    dim3 threads = dim3(32, 32);

    convolution_backward_kernel<<<blocks, threads>>>(y_h,
            filters, vbias, y_v_probs, y_v, input_size, left_upper_padding,
            channel_num, feature_map_size, filter_num, filter_size);
    hipDeviceSynchronize();
}

void CRBM::GPU_compute_d_w(float *v, float *h, float *dw, bool is_init){
    dim3 blocks = dim3(channel_num * filter_num * feature_map_size / 32, 
            input_num * feature_map_size / 32);
    dim3 threads = dim3(filter_size, filter_size);

    compute_d_w_kernel<<<blocks, threads>>>(v, h, dw, is_init, input_size, left_upper_padding,
            channel_num, filter_num, filter_size, feature_map_size);
    hipDeviceSynchronize();
}

void CRBM::start(){
    struct timeval _start_time, _end_time;

    timeFunc(this->CPU_convolution_forward(this->CPU_input->get_data(),
            this->CPU_filters->get_data(), this->CPU_y_h->get_data(),
            this->CPU_hbias->get_data()), "CPU convolutional forward");
    
    timeFunc(this->CPU_max_pooling(this->CPU_y_h->get_data(),
            this->CPU_y_h_probs->get_data(), this->CPU_y_p->get_data()), 
            "CPU max pooling");

    timeFunc(this->CPU_convolution_backward(this->CPU_y_h_probs->get_data(),
            this->CPU_filters->get_data(), this->CPU_vbias->get_data(),
            this->CPU_y_v_probs->get_data(), this->CPU_y_v->get_data()), 
            "CPU convolutional backward");

    timeFunc(this->CPU_compute_d_w(this->CPU_input->get_data(),
            this->CPU_y_h_probs->get_data(), this->CPU_d_w->get_data(),
            true), "CPU compute dw positive phase");

    timeFunc(this->GPU_convolution_forward(this->GPU_input->get_data(),
            this->GPU_filters->get_data(), this->GPU_y_h->get_data(),
            this->GPU_hbias->get_data()), "GPU convolutional forward");
    
    timeFunc(this->GPU_max_pooling(this->GPU_y_h->get_data(),
            this->GPU_y_h_probs->get_data(), this->GPU_y_p->get_data()), 
            "GPU max pooling");
    
    timeFunc(this->GPU_convolution_backward(this->GPU_y_h_probs->get_data(),
            this->GPU_filters->get_data(), this->GPU_vbias->get_data(),
            this->GPU_y_v_probs->get_data(), this->GPU_y_v->get_data()), 
            "GPU convolutional backward");

    timeFunc(this->GPU_compute_d_w(this->GPU_input->get_data(),
            this->GPU_y_h_probs->get_data(), this->GPU_d_w->get_data(),
            true), "GPU compute dw positive phase");

    Matrix* tmp_y_h_probs = new Matrix(this->CPU_y_h_probs->get_row_num(),
                                 this->CPU_y_h_probs->get_col_num());
    this->GPU_y_h_probs->assign(*tmp_y_h_probs);
    this->CPU_y_h_probs->equal_value(*tmp_y_h_probs);
    delete tmp_y_h_probs;

    Matrix* tmp_y_v_probs = new Matrix(this->CPU_y_v_probs->get_row_num(),
                                 this->CPU_y_v_probs->get_col_num());
    this->GPU_y_v_probs->assign(*tmp_y_v_probs);
    this->CPU_y_v_probs->equal_value(*tmp_y_v_probs);
    delete tmp_y_v_probs;

    Matrix* tmp_d_w = new Matrix(this->CPU_d_w->get_row_num(),
                                 this->CPU_d_w->get_col_num());
    this->GPU_d_w->assign(*tmp_d_w);
    this->CPU_d_w->equal_value(*tmp_d_w);
    delete tmp_d_w;
}
